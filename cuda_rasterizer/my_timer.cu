
#include <hip/hip_runtime.h>
#include <chrono>
#include <unordered_map>
#include <string>
#include <algorithm>
#include <iostream>
#include <vector>
#include <utility>
#include "stdio.h"
#include "stdlib.h"


class MyTimer {
public:
    void start(const std::string& name) {
        time_points[name] = std::chrono::high_resolution_clock::now();
    }

    void stop(const std::string& name) {
        auto end_time = std::chrono::high_resolution_clock::now();
        durations[name] += std::chrono::duration<double, std::milli>(end_time - time_points[name]).count();
        count[name] += 1;
    }

    double elapsedMilliseconds(const std::string& name, const std::string& mode = "average") const {
        if (mode == "average")
            return durations.at(name)/count.at(name);
        else 
            return durations.at(name);
    }

    void printAllTimes(
        int iteration,
        int world_size,
        int local_rank,
        const char* log_folder = nullptr
    ) const {
        char* prefix = new char[100];
		sprintf(prefix, "\nit=%d,ws:%d,rk=%d  -->\n", iteration, world_size, local_rank);
		char* filename = new char[100];
		sprintf(filename, "%s/time_ws=%d_rk=%d.log", log_folder, world_size, local_rank);

        std::vector<std::pair<std::string, double>> sortedTimes(durations.begin(), durations.end());
        std::sort(sortedTimes.begin(), sortedTimes.end(), 
            [](const auto& a, const auto& b) {
                return a.first < b.first;
            }
        );
        std::cout << prefix << std::endl;
        for (const auto& pair : sortedTimes) {
            std::cout << pair.first << " time: " << elapsedMilliseconds(pair.first, "sum") << " ms" << std::endl;
        }
        //save in file
        FILE *fp;
        fp = fopen(filename, "a");
        fprintf(fp, "%s", prefix);
        for (const auto& pair : sortedTimes) {
            fprintf(fp, "%s time: %f ms\n", pair.first.c_str(), elapsedMilliseconds(pair.first, "sum"));
        }
        //clean up
        fclose(fp);
        delete[] prefix;
        delete[] filename;
    }

private:
    std::unordered_map<std::string, std::chrono::time_point<std::chrono::high_resolution_clock>> time_points;
    std::unordered_map<std::string, double> durations;
    std::unordered_map<std::string, int> count;
};