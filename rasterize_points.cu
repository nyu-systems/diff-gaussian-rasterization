#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include "cuda_rasterizer/auxiliary.h"
#include <fstream>
#include <string>
#include <functional>

#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}


/////////////////////////////// Preprocess ///////////////////////////////

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
PreprocessGaussiansCUDA(
	const torch::Tensor& means3D,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const torch::Tensor& sh,
    const torch::Tensor& opacity,//3dgs' parametes.
	const float scale_modifier,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,//raster_settings
	const bool debug,
	const pybind11::dict &args) {

	if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
		AT_ERROR("means3D must have dimensions (num_points, 3)");
	}

	const int P = means3D.size(0);
	const int H = image_height;
	const int W = image_width;

	// of shape (P, 2). means2D is (P, 2) in cuda. It will be converted to (P, 3) when is sent back to python to meet torch graph's requirement.
	torch::Tensor means2D = torch::full({P, 2}, 0.0, means3D.options());//TODO: what about require_grads?
	// of shape (P)
	torch::Tensor depths = torch::full({P}, 0.0, means3D.options());
	// of shape (P)
	torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
	// of shape (P, 6)
	torch::Tensor cov3D = torch::full({P, 6}, 0.0, means3D.options());
	// of shape (P, 4)
	torch::Tensor conic_opacity = torch::full({P, 4}, 0.0, means3D.options());
	// of shape (P, 3)
	torch::Tensor rgb = torch::full({P, 3}, 0.0, means3D.options());
	// of shape (P)
	torch::Tensor clamped = torch::full({P, 3}, false, means3D.options().dtype(at::kBool));
	//TODO: compare to original GeometryState implementation, this one does not explicitly do gpu memory alignment. 
	//That may lead to problems. However, pytorch does implicit memory alignment.

	int rendered = 0;//TODO: I could compute rendered here by summing up geomState.tiles_touched. 
	if(P != 0)
	{
		int M = 0;
		if(sh.size(0) != 0)
		{
			M = sh.size(1);
		}

		rendered = CudaRasterizer::Rasterizer::preprocessForward(
			reinterpret_cast<float2*>(means2D.contiguous().data<float>()),//TODO: check whether it supports float2?
			depths.contiguous().data<float>(),
			radii.contiguous().data<int>(),
			cov3D.contiguous().data<float>(),
			reinterpret_cast<float4*>(conic_opacity.contiguous().data<float>()),
			rgb.contiguous().data<float>(),
			clamped.contiguous().data<bool>(),
			P, degree, M,
			W, H,
			means3D.contiguous().data<float>(),
			scales.contiguous().data_ptr<float>(),
			rotations.contiguous().data_ptr<float>(),
			sh.contiguous().data_ptr<float>(),
			opacity.contiguous().data<float>(), 
			scale_modifier,
			viewmatrix.contiguous().data<float>(), 
			projmatrix.contiguous().data<float>(),
			campos.contiguous().data<float>(),
			tan_fovx,
			tan_fovy,
			prefiltered,
			debug,
			args);
	}
	return std::make_tuple(rendered, means2D, depths, radii, cov3D, conic_opacity, rgb, clamped);
}


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
  PreprocessGaussiansBackwardCUDA(
	const torch::Tensor& radii,
	const torch::Tensor& cov3D,
	const torch::Tensor& clamped,//the above are all per-Gaussian intemediate results.
	const torch::Tensor& means3D,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const torch::Tensor& sh,//input of this operator
	const float scale_modifier,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const int degree,
	const torch::Tensor& campos,//rasterization setting.
	const torch::Tensor& dL_dmeans2D,// (P, 3)
	const torch::Tensor& dL_dconic_opacity,
	const torch::Tensor& dL_dcolors,//gradients of output of this operator
	const int R,
	const bool debug,
	const pybind11::dict &args)
{
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  // set dL_dconic[..., 0, 0] = dL_dconic_opacity[..., 0]
  dL_dconic.select(1, 0).select(1, 0).copy_(dL_dconic_opacity.select(1, 0));// select() is kind of view, it does not allocate new memory.
  // set dL_dconic[..., 0, 1] = dL_dconic_opacity[..., 1]
  dL_dconic.select(1, 0).select(1, 1).copy_(dL_dconic_opacity.select(1, 1));
  // set dL_dconic[..., 1, 1] = dL_dconic_opacity[..., 2]
  dL_dconic.select(1, 1).select(1, 1).copy_(dL_dconic_opacity.select(1, 2));
  dL_dconic = dL_dconic.contiguous();
  //TODO: is this correct usage?

  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  // set dL_dopacity[..., 0] = dL_dconic_opacity[..., 3]
  dL_dopacity.select(1, 0).copy_(dL_dconic_opacity.select(1, 3));
  dL_dopacity = dL_dopacity.contiguous();

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  //dL_dcov3D is itermidiate result to compute dL_drotations and dL_dscales, do not need to return to python.
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());

  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::preprocessBackward(
		radii.contiguous().data<int>(),
		cov3D.contiguous().data<float>(),
		clamped.contiguous().data<bool>(),//the above are all per-Gaussian intermediate results.
		P, degree, M, R,
		W, H, //rasterization setting.
		means3D.contiguous().data<float>(),
		scales.data_ptr<float>(),
  	    rotations.data_ptr<float>(),
		sh.contiguous().data<float>(),//input of this operator
		scale_modifier,
		viewmatrix.contiguous().data<float>(),
	    projmatrix.contiguous().data<float>(),
	    campos.contiguous().data<float>(),
	    tan_fovx,
	    tan_fovy,//rasterization setting.
	    dL_dmeans2D.contiguous().data<float>(),
	    dL_dconic.contiguous().data<float>(),
	    dL_dcolors.contiguous().data<float>(),//gradients of output of this operator
	    dL_dmeans3D.contiguous().data<float>(),
	    dL_dcov3D.contiguous().data<float>(),
	    dL_dscales.contiguous().data<float>(),
	    dL_drotations.contiguous().data<float>(),
	    dL_dsh.contiguous().data<float>(),//gradients of input of this operator
		debug,
		args);
  }

  return std::make_tuple(dL_dmeans3D, dL_dscales, dL_drotations, dL_dsh, dL_dopacity);
}


////////////////////// GetDistributionStrategy ////////////////////////

torch::Tensor GetDistributionStrategyCUDA(
    const int image_height,
    const int image_width,// image setting
	torch::Tensor& means2D,// (P, 2)
	torch::Tensor& radii,
	const bool debug,
	const pybind11::dict &args)
{
	const int P = means2D.size(0);
	const int TILE_Y = (image_height + BLOCK_Y - 1) / BLOCK_Y;
	const int TILE_X = (image_width + BLOCK_X - 1) / BLOCK_X;
	
	torch::Tensor compute_locally = torch::full({TILE_Y, TILE_X}, false, means2D.options().dtype(at::kBool).requires_grad(false));

	torch::Device device(torch::kCUDA);
	torch::TensorOptions options(torch::kByte);
	torch::Tensor distBuffer = torch::empty({0}, options.device(device));
	std::function<char*(size_t)> distFunc = resizeFunctional(distBuffer);

	if (P != 0)
	{
		CudaRasterizer::Rasterizer::getDistributionStrategy(
			distFunc,
			P,
			image_width, image_height,
			reinterpret_cast<float2*>(means2D.contiguous().data<float>()),
			radii.contiguous().data<int>(),
			compute_locally.contiguous().data<bool>(),
			debug,
			args);
	}
	return compute_locally;
}


/////////////////////////////// Render ///////////////////////////////

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RenderGaussiansCUDA(
	const torch::Tensor& background,
    const int image_height,
    const int image_width,// image setting
	torch::Tensor& means2D,// (P, 2)
	torch::Tensor& depths,
	torch::Tensor& radii,
	torch::Tensor& conic_opacity,
	torch::Tensor& rgb,//3dgs intermediate results
	const torch::Tensor& compute_locally,
	const bool debug,
	const pybind11::dict &args)
{
  const int P = means2D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means2D.options().dtype(torch::kInt32);
  auto float_opts = means2D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);

  const int TILE_Y = (H + BLOCK_Y - 1) / BLOCK_Y;
  const int TILE_X = (W + BLOCK_X - 1) / BLOCK_X;
  const int tile_num = TILE_Y * TILE_X;
  torch::Tensor n_render = torch::full({tile_num}, 0, int_opts);
  torch::Tensor n_consider = torch::full({tile_num}, 0, int_opts);
  torch::Tensor n_contrib = torch::full({tile_num}, 0, int_opts);

  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  rendered = CudaRasterizer::Rasterizer::renderForward(
		geomFunc,
		binningFunc,
		imgFunc,//buffer
	    P,
		background.contiguous().data<float>(),
		W, H,//image setting
		reinterpret_cast<float2*>(means2D.contiguous().data<float>()),
		depths.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		reinterpret_cast<float4*>(conic_opacity.contiguous().data<float>()),
		rgb.contiguous().data<float>(),//3dgs intermediate results
		compute_locally.contiguous().data<bool>(),
		out_color.contiguous().data<float>(),
		n_render.contiguous().data<int>(),
		n_consider.contiguous().data<int>(),
		n_contrib.contiguous().data<int>(),//output
		debug,
		args);
  }
  return std::make_tuple(rendered, out_color, n_render, n_consider, n_contrib, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
RenderGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const int R,
	const torch::Tensor& geomBuffer,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const torch::Tensor& compute_locally,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& means2D,// (P, 2)
	const torch::Tensor& conic_opacity,
	const torch::Tensor& rgb,
	const bool debug,
	const pybind11::dict &args)
{
  const int P = means2D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means2D.options());//TODO: does options for a tensor and its grad differ from each other?
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means2D.options());//if we use mixed precision, dtype in options() is different now. If we also do swapping, device could be different. 
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means2D.options());//The requires_grad property for the gradient tensor is typically False
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means2D.options());

  if(P != 0)
  {
	  CudaRasterizer::Rasterizer::renderBackward(
		P, R,
		background.contiguous().data<float>(),
		W, H,//rasterization settings.  
		reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),//buffer that contains intermedia results
		compute_locally.contiguous().data<bool>(),
		dL_dout_color.contiguous().data<float>(),//gradient of output
		dL_dmeans2D.contiguous().data<float>(),
		dL_dconic.contiguous().data<float>(),
		dL_dopacity.contiguous().data<float>(),
		dL_dcolors.contiguous().data<float>(),//gradient of inputs
		reinterpret_cast<float2*>(means2D.contiguous().data<float>()),
		reinterpret_cast<float4*>(conic_opacity.contiguous().data<float>()),
		rgb.contiguous().data<float>(),
		debug,
		args);
  }

  torch::Tensor dL_dconic_opacity = torch::zeros({P, 4}, means2D.options());
  // set dL_dconic_opacity[..., 0] = dL_dconic[..., 0, 0]
  dL_dconic_opacity.select(1, 0).copy_(dL_dconic.select(1, 0).select(1, 0));
  // set dL_dconic_opacity[..., 1] = dL_dconic[..., 0, 1]
  dL_dconic_opacity.select(1, 1).copy_(dL_dconic.select(1, 0).select(1, 1));
  // set dL_dconic_opacity[..., 2] = dL_dconic[..., 1, 1]
  dL_dconic_opacity.select(1, 2).copy_(dL_dconic.select(1, 1).select(1, 1));
  // set dL_dconic_opacity[..., 3] = dL_dopacity[..., 0]
  dL_dconic_opacity.select(1, 3).copy_(dL_dopacity.select(1, 0));
  
  //TODO: in pytorch, when the reference to a tensor decreases to 0, the memory will be freed.
  //But what will happen to libtorch?
  return std::make_tuple(dL_dmeans2D, dL_dconic_opacity, dL_dcolors);
}

/////////////////////////////// Loss ///////////////////////////////

std::tuple<torch::Tensor, torch::Tensor>
FusedL1LossCUDA(
  torch::Tensor& image,
  torch::Tensor& gt_image,
  torch::Tensor& mask,
  float lambda_dssim
)
{
  int C = image.size(0);
  int H = image.size(1);
  int W = image.size(2);
  torch::Tensor dL_dimage = torch::zeros({C, H, W}, image.options());
  auto options = torch::TensorOptions().device(torch::kCUDA);
  torch::Tensor loss = torch::zeros({}, options);

  CudaRasterizer::Rasterizer::l1lossForwardBackward(
    image.contiguous().data<float>(),
    gt_image.contiguous().data<float>(),
    mask.contiguous().data<bool>(),
    C, H, W,
    lambda_dssim,
    loss.contiguous().data<float>(),
    dL_dimage.contiguous().data<float>()
  );
  
  return std::make_tuple(loss, dL_dimage);
}

/////////////////////////////// Utility tools ///////////////////////////////

__global__ void getTouchedIdsBool(
	int P,
	int height,
	int width,
	int world_size,
	const float2* means2D,
	const int* radii,// NOTE: radii is not const in getRect()
	const int* dist_global_strategy,
	bool* touchedIdsBool,
	bool avoid_pixel_all2all)
{
	auto i = cg::this_grid().thread_rank();
	if (i < P)
	{
		uint2 rect_min, rect_max;
		dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);

		getRect(means2D[i], radii[i], rect_min, rect_max, tile_grid);
		
		// method 1:
		int touched_min_tile_idx = rect_min.y * tile_grid.x + rect_min.x;
		int touched_max_tile_idx = (rect_max.y - 1 ) * tile_grid.x + rect_max.x - 1;

		if ( touched_max_tile_idx < touched_min_tile_idx )
			return;
			
		for (int rk = 0; rk < world_size; rk++)
		{
			int tile_l = *(dist_global_strategy+rk);
			int tile_r = *(dist_global_strategy+rk+1);
			if (avoid_pixel_all2all) {
				// we could avoid the pixel all2all by rendering the pixels that are near border and out of border. 
				tile_l -= tile_grid.x+1;
				tile_r += tile_grid.x+1;
			}

			if (touched_max_tile_idx < tile_l || touched_min_tile_idx >= tile_r)
				continue;
			
			// TODO: If one worker's tiles are fewer than one row, then it is buggy. 
			// If we have other workload_division dimension, then we need to change this. 
			touchedIdsBool[i * world_size + rk] = true;
		}
		

		
	}
}

torch::Tensor GetLocal2jIdsBoolCUDA(
	int image_height,
	int image_width,
	int local_rank,
	int world_size,
	const torch::Tensor& means2D,
	const torch::Tensor& radii,
	const torch::Tensor& dist_global_strategy,
	const pybind11::dict &args)
{	
	const int P = means2D.size(0);
	const int H = image_height;
	const int W = image_width;
	bool avoid_pixel_all2all = args["avoid_pixel_all2all"].cast<bool>();

	torch::Tensor local2jIdsBool = torch::full({P, world_size}, false, means2D.options().dtype(torch::kBool));

	getTouchedIdsBool << <(P + ONE_DIM_BLOCK_SIZE - 1) / ONE_DIM_BLOCK_SIZE, ONE_DIM_BLOCK_SIZE >> >(
		P,
		H,
		W,
		world_size,
		reinterpret_cast<float2*>(means2D.contiguous().data<float>()),
		radii.contiguous().data<int>(),
		dist_global_strategy.contiguous().data<int>(),
		local2jIdsBool.contiguous().data<bool>(),
		avoid_pixel_all2all
	);

	return local2jIdsBool;
}








////////////////////// Image Distribution Utilities ////////////////////////


__global__ void get_touched_locally(
	const int tile_num,
	const int TILE_Y,
	const int TILE_X,
	const bool* compute_locally,
	bool* touched_locally
) {
	auto i = cg::this_grid().thread_rank();
	if (i < tile_num && compute_locally[i])
	{
		int y = i / TILE_X;
		int x = i % TILE_X;
		touched_locally[i] = true;
		const int dx[8] = {-1, -1, -1, 0, 0, 1, 1, 1};//by default, extension_distance is 1.
		const int dy[8] = {-1, 0, 1, -1, 1, -1, 0, 1};
		for (int k = 0; k < 8; k++)
		{
			int ny = y + dy[k];
			int nx = x + dx[k];
			if (ny >= 0 && ny < TILE_Y && nx >= 0 && nx < TILE_X)
				touched_locally[ny * TILE_X + nx] = true;
		}
	}
}

torch::Tensor GetTouchedLocally(
	const torch::Tensor& compute_locally,
	const int image_height,
	const int image_width,
	const int extension_distance
) {
	const int TILE_Y = (image_height + BLOCK_Y - 1) / BLOCK_Y;
	const int TILE_X = (image_width + BLOCK_X - 1) / BLOCK_X;
	const int tile_num = TILE_Y * TILE_X;// NOTE: at most, we have 5000*5000/16/16 = 97656 tiles
	
	torch::Tensor touched_locally = torch::full({TILE_Y, TILE_X}, false, compute_locally.options());

	get_touched_locally<<< (tile_num + ONE_DIM_BLOCK_SIZE - 1) / ONE_DIM_BLOCK_SIZE, ONE_DIM_BLOCK_SIZE >>> (
		tile_num,
		TILE_Y,
		TILE_X,
		compute_locally.contiguous().data<bool>(),
		touched_locally.contiguous().data<bool>()
	);
	return touched_locally;
}


__global__ void load_image_tiles_by_pos(
	int N,
	int image_height,
	int image_width,
	int min_pixel_y,
	int min_pixel_x,
	int local_image_rect_height,
	int local_image_rect_width,
	const int64_t* all_tiles_pos,
	const float* local_image_rect,
	float* image_tiles)
{
	auto block = cg::this_thread_block();
	int i = block.group_index().x;
	int tile_pos_y = (int)all_tiles_pos[ i * 2 ];
	int tile_pos_x = (int)all_tiles_pos[ i * 2 + 1 ];

	int image_x = tile_pos_x * BLOCK_X + block.thread_index().x;
	int image_y = tile_pos_y * BLOCK_Y + block.thread_index().y;

	int image_tiles_offset = i * 3 * BLOCK_X * BLOCK_Y + block.thread_rank();
	int tile_pixels_num = BLOCK_X * BLOCK_Y;

	if (image_x < image_width && image_y < image_height)
	{
		int local_image_rect_x = image_x - min_pixel_x;
		int local_image_rect_y = image_y - min_pixel_y;
		int local_image_rect_offset = local_image_rect_y * local_image_rect_width + local_image_rect_x;
		int local_image_rect_pixels_num = local_image_rect_height * local_image_rect_width;

		image_tiles[image_tiles_offset] = local_image_rect[local_image_rect_offset];
		image_tiles[image_tiles_offset + tile_pixels_num] = local_image_rect[local_image_rect_offset + local_image_rect_pixels_num];
		image_tiles[image_tiles_offset + 2 * tile_pixels_num] = local_image_rect[local_image_rect_offset + 2 * local_image_rect_pixels_num];
	}
	else
	{
		image_tiles[image_tiles_offset] = 0.0;
		image_tiles[image_tiles_offset + tile_pixels_num] = 0.0;
		image_tiles[image_tiles_offset + 2 * tile_pixels_num] = 0.0;
	}
}

__global__ void set_image_tiles_by_pos(
	int N,
	int image_height,
	int image_width,
	int min_pixel_y,
	int min_pixel_x,
	int local_image_rect_height,
	int local_image_rect_width,
	const int64_t* all_tiles_pos,
	float* local_image_rect,
	const float* image_tiles)
{
	auto block = cg::this_thread_block();
	int i = block.group_index().x;
	int tile_pos_y = (int)all_tiles_pos[ i * 2 ];
	int tile_pos_x = (int)all_tiles_pos[ i * 2 + 1 ];

	int image_x = tile_pos_x * BLOCK_X + block.thread_index().x;
	int image_y = tile_pos_y * BLOCK_Y + block.thread_index().y;

	int image_tiles_offset = i * 3 * BLOCK_X * BLOCK_Y + block.thread_rank();
	if (image_x < image_width && image_y < image_height)
	{
		int local_image_rect_x = image_x - min_pixel_x;
		int local_image_rect_y = image_y - min_pixel_y;
		int local_image_rect_offset = local_image_rect_y * local_image_rect_width + local_image_rect_x;
		int local_image_rect_pixels_num = local_image_rect_height * local_image_rect_width;
		int tile_pixels_num = BLOCK_X * BLOCK_Y;

		local_image_rect[local_image_rect_offset] = image_tiles[image_tiles_offset];
		local_image_rect[local_image_rect_offset + local_image_rect_pixels_num] = image_tiles[image_tiles_offset + tile_pixels_num];
		local_image_rect[local_image_rect_offset + 2 * local_image_rect_pixels_num] = image_tiles[image_tiles_offset + 2 * tile_pixels_num];
	}
}

torch::Tensor LoadImageTilesByPos(
	const torch::Tensor& local_image_rect,
	const torch::Tensor& all_tiles_pos,
	int image_height,
	int image_width,
	int min_pixel_y,
	int min_pixel_x,
	int local_image_rect_height,
	int local_image_rect_width)
{
	const int N = all_tiles_pos.size(0);
	dim3 tile_grid(N, 1, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	torch::Tensor image_tiles = torch::full({N, 3, BLOCK_Y, BLOCK_X}, 0.0, local_image_rect.options());
	// if image.options() requires_grad, then image_tiles.options() requires_grad should also requires_grad.

	load_image_tiles_by_pos<<< tile_grid, block >>>(
		N,
		image_height,
		image_width,
		min_pixel_y,
		min_pixel_x,
		local_image_rect_height,
		local_image_rect_width,
		all_tiles_pos.contiguous().data<int64_t>(),
		local_image_rect.contiguous().data<float>(),
		image_tiles.contiguous().data<float>()
	);
	return image_tiles;
}

torch::Tensor SetImageTilesByPos(
	const torch::Tensor& all_tiles_pos,
	const torch::Tensor& image_tiles,
	int image_height,
	int image_width,
	int min_pixel_y,
	int min_pixel_x,
	int local_image_rect_height,
	int local_image_rect_width)
{
	const int N = all_tiles_pos.size(0);
	dim3 tile_grid(N, 1, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	torch::Tensor local_image_rect = torch::full({3, local_image_rect_height, local_image_rect_width}, 0.0, image_tiles.options());

	set_image_tiles_by_pos <<< tile_grid, block >>>(
		N,
		image_height,
		image_width,
		min_pixel_y,
		min_pixel_x,
		local_image_rect_height,
		local_image_rect_width,
		all_tiles_pos.contiguous().data<int64_t>(),
		local_image_rect.contiguous().data<float>(),
		image_tiles.contiguous().data<float>()
	);
	return local_image_rect;
}


__global__ void get_pixels_compute_locally_and_in_rect(
	int image_height,
	int image_width,
	int local_image_height,
	int local_image_width,
	int min_pixel_y,
	int min_pixel_x,
	const bool* compute_locally,
	bool* pixels_compute_locally_and_in_rect)
{
	auto block = cg::this_thread_block();
	int local_pixel_x = block.group_index().x * BLOCK_X + block.thread_index().x;
	int local_pixel_y = block.group_index().y * BLOCK_Y + block.thread_index().y;

	if (local_pixel_x < local_image_width && local_pixel_y < local_image_height)
	{
		int global_pixel_x = local_pixel_x + min_pixel_x;
		int global_pixel_y = local_pixel_y + min_pixel_y;
		int global_tile_x = global_pixel_x / BLOCK_X;
		int global_tile_y = global_pixel_y / BLOCK_Y;
		int TILE_X = (image_width + BLOCK_X - 1) / BLOCK_X;
		pixels_compute_locally_and_in_rect[local_pixel_y * local_image_width + local_pixel_x] = compute_locally[global_tile_y * TILE_X + global_tile_x];
	}
}

torch::Tensor GetPixelsComputeLocallyAndInRect(
	const torch::Tensor& compute_locally,
	int image_height,
	int image_width,
	int min_pixel_y,
	int max_pixel_y,
	int min_pixel_x,
	int max_pixel_x)
{
	int local_image_height = max_pixel_y - min_pixel_y;
	int local_image_width = max_pixel_x - min_pixel_x;
	const int TILE_Y = (local_image_height + BLOCK_Y - 1) / BLOCK_Y;
	const int TILE_X = (local_image_width + BLOCK_X - 1) / BLOCK_X;

	dim3 tile_grid(TILE_X, TILE_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);
	
	torch::Tensor pixels_compute_locally_and_in_rect = torch::full({max_pixel_y - min_pixel_y, max_pixel_x - min_pixel_x}, false, compute_locally.options().dtype(at::kBool));

	get_pixels_compute_locally_and_in_rect << < tile_grid, block >> > (
		image_height,
		image_width,	
		local_image_height,
		local_image_width,
		min_pixel_y,
		min_pixel_x,
		compute_locally.contiguous().data<bool>(),
		pixels_compute_locally_and_in_rect.contiguous().data<bool>()
	);
	return pixels_compute_locally_and_in_rect;
}

std::tuple<int, int, int> GetBlockXY()
{
	return std::make_tuple(BLOCK_X, BLOCK_Y, ONE_DIM_BLOCK_SIZE);
}